#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "kernel.h"
#include "../common/file.h"
#include "../common/errorManager.h"




int main(int argc, char const *argv[]){

    float *A_h, *B_h, *C_h, *A_d, *B_d, *C_d;
    unsigned int vec_size;
    char* infileA = "input/inputA.dat";
    char* infileB = "input/inputB.dat";
    char* outfile = "output/out.dat";
    size_t vec_bytes;
    
    dim3 dim_grid, dim_block;
    hipError_t err;

    //1. Allocate host memory for the input/output vectors
    readVector(infileA, &A_h, &vec_size);
    readVector(infileB, &B_h, &vec_size);
    vec_bytes = sizeof(float) * vec_size;
    C_h = (float*) malloc(vec_bytes);
    printf("Load vectors with size %i (bytes %d)\n", vec_size, vec_bytes);

    //2. Allocate device memory for the input/output vectors
    err = hipMalloc(&A_d, vec_bytes);
    checkCudaError(err);
    err = hipMalloc(&B_d, vec_bytes);
    checkCudaError(err);
    err = hipMalloc(&C_d, vec_bytes);
    checkCudaError(err);

    //3. Copy the input vectors from the host memory to the device memory
    err = hipMemcpy(A_d, A_h, vec_bytes, hipMemcpyHostToDevice);
    checkCudaError(err);
    err = hipMemcpy(B_d, B_h, vec_bytes, hipMemcpyHostToDevice);
    checkCudaError(err);
    err = hipMemset(C_d, 0, vec_bytes);
    checkCudaError(err);

    //4. Initialize thread block and kernel grid dimensions
    dim_grid = dim3(ceil(vec_size/512.0));
    dim_block = dim3(512);

    //Invoke CUDA kernel
    vecAdd<<<dim_grid, dim_block>>>(C_d, A_d, B_d, vec_size);
    checkCudaError(hipGetLastError());

    //Copy the result back to the host & print result
    err = hipMemcpy(C_h, C_d, vec_bytes, hipMemcpyDeviceToHost);
    writeVector(outfile, C_h, vec_size);

    //Free device memory allocations
    hipFree(A_d); hipFree(B_d); hipFree(C_d);
    free(A_h); free(A_h); free(A_h);

    return 0;
}


