#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "kernel.h"
#include "../common/file.h"

void checkCudaError(const hipError_t err);

int main(int argc, char const *argv[]){

    float *A_h, *B_h, *C_h, *A_d, *B_d, *C_d;
    unsigned int vec_size;
    char infileA[] = "inputs/input1.dat";
    char infileB[] = "inputs/input2.dat";
    char outfile[] = "output/out.dat";
    size_t vec_bytes;
    
    dim3 dim_grid, dim_block;
    hipError_t err;

    //1. Allocate host memory for the input/output vectors
    readVector(infileA, &A_h, &vec_size);
    readVector(infileB, &B_h, &vec_size);
    vec_bytes = sizeof(float) * vec_size;
    C_h = (float*) malloc(vec_bytes);
    printf("Load vectors with size %i (bytes %lu)\n", vec_size, vec_bytes);

    //2. Allocate device memory for the input/output vectors
    err = hipMalloc(&A_d, vec_bytes);
    checkCudaError(err);
    err = hipMalloc(&B_d, vec_bytes);
    checkCudaError(err);
    err = hipMalloc(&C_d, vec_bytes);
    checkCudaError(err);

    //3. Copy the input vectors from the host memory to the device memory
    err = hipMemcpy(A_d, A_h, vec_bytes, hipMemcpyHostToDevice);
    checkCudaError(err);
    err = hipMemcpy(B_d, B_h, vec_bytes, hipMemcpyHostToDevice);
    checkCudaError(err);
    err = hipMemset(C_d, 0, vec_bytes);
    checkCudaError(err);

    //4. Initialize thread block and kernel grid dimensions
    dim_grid = dim3(ceil(vec_size/512.0));
    dim_block = dim3(512);

    //Invoke CUDA kernel
    vecAdd<<<dim_grid, dim_block>>>(C_d, A_d, B_d, vec_size);
    checkCudaError(hipGetLastError());

    //Copy the result back to the host & print result
    err = hipMemcpy(C_h, C_d, vec_bytes, hipMemcpyDeviceToHost);
    writeVectorFloat(outfile, C_h, vec_size);

    //Free device memory allocations
    hipFree(A_d); hipFree(B_d); hipFree(C_d);
    free(A_h); free(B_h); free(C_h);

    return 0;
}


void checkCudaError(const hipError_t err){
    if(err == hipSuccess) return;
    printf("%s : %s", hipGetErrorName(err), hipGetErrorString(err));
    exit(4);
}